
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime_api.h"

typedef float2 Complex;

void
PerformCUDAFFT(float *inputData, float *outputData, unsigned int numSamples) {
    hipfftHandle plan;
    hipfftComplex *inputDataG, *outputDataG;
    int i;

    float *inputDataC, *outputDataC;
    outputDataC = (float*) malloc(sizeof(float) * numSamples * 2);
    inputDataC  = (float*) malloc(sizeof(float) * numSamples * 2);

    for (i = 0; i < numSamples; i++) {
      inputDataC[i*2]     = inputData[i];
      inputDataC[i*2 + 1] = 0.0f;
    }

    hipMalloc((void**)&inputDataG,  sizeof(hipfftComplex)*numSamples);
    hipMalloc((void**)&outputDataG, sizeof(hipfftComplex)*numSamples);

    hipMemcpy(inputDataG, inputDataC, sizeof(hipfftComplex)*numSamples, hipMemcpyHostToDevice);
    hipfftPlan1d(&plan, numSamples, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, inputDataG, outputDataG, HIPFFT_FORWARD);
    hipfftDestroy(plan);
    hipMemcpy(outputDataC, outputDataG, sizeof(hipfftComplex)*numSamples, hipMemcpyDeviceToHost);
    hipFree(inputDataG);
    hipFree(outputDataG);
    
    for (i = 0; i < numSamples; i++) {
        outputData[i] = outputDataC[i * 2];
    }
    free(outputDataC);
    free(inputDataC);
}
